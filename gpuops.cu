#include "hip/hip_runtime.h"
#include "gpuops.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <sys/resource.h>

#define INT_M 2

const int N = 20 * (1 << 20);

double get_time()
{
	struct timeval t;
	struct timezone tzp;
	gettimeofday(&t, &tzp);
	return t.tv_sec + t.tv_usec*1e-6;
}

//float operations
__global__ void add_float(int n, float a, float *x, float *y, float *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
	y[i] = a*x[i] + y[i];//y=4
	z[i] = a*x[i] + z[i];//z=5
	z[i] = a*y[i] + z[i];//z=13
	y[i] = a*x[i] + z[i];//y=15
        y[i] = a*x[i] + y[i];//y=17
	}
}

//int operations
__global__ void add_int(int n, int a, int *x, int *y, int *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
	y[i] = a*x[i] + y[i];//y=4
	z[i] = a*x[i] + z[i];//z=5
	z[i] = a*y[i] + z[i];//z=13
        y[i] = a*x[i] + z[i];//y=15
        y[i] = a*x[i] + y[i];//y=17
	}
}

float speed_test_int(int n_blocks, int n_cores){
	int *x, *y, *z, *d_x, *d_y, *d_z;
	x = (int*)malloc(N*sizeof(int));
	y = (int*)malloc(N*sizeof(int));
	z = (int*)malloc(N*sizeof(int));

	hipMalloc(&d_x, N*sizeof(int)); 
	hipMalloc(&d_y, N*sizeof(int));
	hipMalloc(&d_z, N*sizeof(int));

	for (int i = 0; i < N; i++) {
		x[i] = 1;
		y[i] = 2;
		z[i] = 3;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_z, z, N*sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start);

	printf("\nPerforming integer operation on %d elements . . .\n", N);
	//blocks, cores
	add_int<<<n_blocks, n_cores>>>(N, 2, d_x, d_y, d_z);

	hipEventRecord(stop);

	hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(z, d_z, N*sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float mseconds = 0;
	hipEventElapsedTime(&mseconds, start, stop);

	int maxError_y = 0;
	int maxError_z = 0;
	for (int i = 0; i < N; i++) {
	maxError_y = max(maxError_y, abs(y[i]-17));
	maxError_z = max(maxError_z, abs(z[i]-13));
	}

	hipFree( x );
        hipFree( y );
        hipFree( z );

//	printf("\nMax error y: %fn", maxError_y);
//	printf("\nMax error z: %fn", maxError_z);
	printf("\nTime elapsed: %f", mseconds);

	return mseconds;
}

float speed_test_float(int n_blocks, int n_cores){
	float *x, *y, *z, *d_x, *d_y, *d_z;
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));
	z = (float*)malloc(N*sizeof(float));

	hipMalloc(&d_x, N*sizeof(float)); 
	hipMalloc(&d_y, N*sizeof(float));
	hipMalloc(&d_z, N*sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
		z[i] = 3.0f;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_z, z, N*sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(start);

	printf("\nPerforming float operation on %d elements . . .\n", N);
	//blocks, cores
	add_float<<<n_blocks, n_cores>>>(N, 2.0f, d_x, d_y, d_z);

	hipEventRecord(stop);

	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(z, d_z, N*sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float mseconds = 0;
	hipEventElapsedTime(&mseconds, start, stop);

	hipFree( x );
	hipFree( y );
	hipFree( z );

	float maxError_y = 0.0f;
	float maxError_z = 0.0f;
	for (int i = 0; i < N; i++) {
	maxError_y = max(maxError_y, abs(y[i]-17.0f));
	maxError_z = max(maxError_z, abs(z[i]-13.0f));
	}

//	printf("\nMax error y: %fn", maxError_y);
//	printf("\nMax error z: %fn", maxError_z);
	printf("\nTime elapsed: %f", mseconds);

	return mseconds;
}


double bandwidth_test(int size){
	int n = size;
	
        char *x, *d_x, *d_y;
        x = (char*)malloc(n*sizeof(char));

        hipMalloc(&d_x, n*sizeof(char));
        hipMalloc(&d_y, n*sizeof(char));

        for (int i = 0; i < n; i++) {
                x[i] = 'a';
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipMemcpy(d_x, x, n*sizeof(char), hipMemcpyHostToDevice);

        hipEventRecord(start);

        printf("\nRuning memory bandwidth test with %d bytes . . .\n", n);

        hipEventRecord(stop);

        hipMemcpy(x, d_x, n*sizeof(char), hipMemcpyDeviceToHost);

        hipEventSynchronize(stop);
        float mseconds = 0;
        hipEventElapsedTime(&mseconds, start, stop);

        hipFree( x );

        printf("\nTime elapsed: %fms", mseconds);
	
	double bandwidth = (n / (mseconds/1000))*1e-9;
        printf("\nBandwidth (GB/s): %f\n", bandwidth);

        return bandwidth;
}

int ConvertSMVer2Cores(int major, int minor)
{
        // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
        typedef struct {
                int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
                int Cores;
        } sSMtoCores;

        sSMtoCores nGpuArchCoresPerSM[] =
        { { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
          { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
          { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
          { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
          { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
          { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
          { 0x30, 192}, // Fermi Generation (SM 3.0) GK10x class
          {   -1, -1 }
        };

        int index = 0;
        while (nGpuArchCoresPerSM[index].SM != -1) {
                if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
                        return nGpuArchCoresPerSM[index].Cores;
                }
                index++;
        }
        printf("MapSMtoCores SM %d.%d is undefined (please update to the latest SDK)!\n", major, minor);
        return -1;
}

TestResult gpu_test() {
	TestResult result;
	int dev =0, n_blocks = 2, n_cores = 384; //blocks per grid and threads per block
	
	//Getting number of GPU cores	
	hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
	n_blocks = deviceProp.multiProcessorCount;
	n_cores = ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount;
	printf("\n  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
	        n_blocks, ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),n_cores);	
	
	//Running tests
	n_blocks = ceil(N+n_cores)/n_cores;	
	
	printf("\n----- Runing GPU speed test - integer operations -----\n");
	result.time_int = speed_test_int(n_blocks, n_cores);	
	printf("\n----- Runing GPU speed test - float operations -----\n");
	result.time_float = speed_test_float((N+511)/n_cores, n_cores);

	printf("\n----- Runing memory tests -----\n");
	result.bandwidth[0] = bandwidth_test(1);
	result.bandwidth[1] = bandwidth_test(1000);
	result.bandwidth[2] = bandwidth_test(1000000);

	//bandwidth test
	return result;
}


